#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2010, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.
\*--------------------------------------------------------------------------*/

//! Bicubic interpolated texture lookup, using unnormalized coordinates.
//! Fast implementation, using 4 trilinear lookups.
//! @param tex  2D texture
//! @param x  unnormalized x texture coordinate
//! @param y  unnormalized y texture coordinate
template<class floatN, class T, enum hipTextureReadMode mode>
__device__ floatN CUBICTEX2D(texture<T, 2, mode> tex, float x, float y)
{
	// transform the coordinate from [0,extent] to [-0.5, extent-0.5]
	const float2 coord_grid = make_float2(x - 0.5f, y - 0.5f);
	const float2 index = floor(coord_grid);
	const float2 fraction = coord_grid - index;
	float2 w0, w1, w2, w3;
	WEIGHTS(fraction, w0, w1, w2, w3);

	const float2 g0 = w0 + w1;
	const float2 g1 = w2 + w3;
	const float2 h0 = (w1 / g0) - make_float2(0.5f) + index;  //h0 = w1/g0 - 1, move from [-0.5, extent-0.5] to [0, extent]
	const float2 h1 = (w3 / g1) + make_float2(1.5f) + index;  //h1 = w3/g1 + 1, move from [-0.5, extent-0.5] to [0, extent]

	// fetch the four linear interpolations
	floatN tex00 = tex2D(tex, h0.x, h0.y);
	floatN tex10 = tex2D(tex, h1.x, h0.y);
	floatN tex01 = tex2D(tex, h0.x, h1.y);
	floatN tex11 = tex2D(tex, h1.x, h1.y);

	// weigh along the y-direction
	tex00 = g0.y * tex00 + g1.y * tex01;
	tex10 = g0.y * tex10 + g1.y * tex11;

	// weigh along the x-direction
	return (g0.x * tex00 + g1.x * tex10);
}


// Specializations

// These specializations fill in the floatN and T class types and therefore
// allow the cubicTex2D function to be called without any template arguments,
// thus with any <> brackets.

// 1-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<float, 2, mode> tex, float x, float y) {return CUBICTEX2D<float, float, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<uchar, 2, mode> tex, float x, float y) {return CUBICTEX2D<float, uchar, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<char, 2, mode> tex, float x, float y) {return CUBICTEX2D<float, char, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<ushort, 2, mode> tex, float x, float y) {return CUBICTEX2D<float, ushort, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<short, 2, mode> tex, float x, float y) {return CUBICTEX2D<float, short, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<uint, 2, mode> tex, float x, float y) {return CUBICTEX2D<float, uint, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<int, 2, mode> tex, float x, float y) {return CUBICTEX2D<float, int, mode>(tex, x, y);}
// 2-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<float2, 2, mode> tex, float x, float y) {return CUBICTEX2D<float2, float2, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<uchar2, 2, mode> tex, float x, float y) {return CUBICTEX2D<float2, uchar2, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<char2, 2, mode> tex, float x, float y) {return CUBICTEX2D<float2, char2, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<ushort2, 2, mode> tex, float x, float y) {return CUBICTEX2D<float2, ushort2, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<short2, 2, mode> tex, float x, float y) {return CUBICTEX2D<float2, short2, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<uint2, 2, mode> tex, float x, float y) {return CUBICTEX2D<float2, uint2, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<int2, 2, mode> tex, float x, float y) {return CUBICTEX2D<float2, int2, mode>(tex, x, y);}
// 3-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<float3, 2, mode> tex, float x, float y) {return CUBICTEX2D<float3, float3, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<uchar3, 2, mode> tex, float x, float y) {return CUBICTEX2D<float3, uchar3, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<char3, 2, mode> tex, float x, float y) {return CUBICTEX2D<float3, char3, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<ushort3, 2, mode> tex, float x, float y) {return CUBICTEX2D<float3, ushort3, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<short3, 2, mode> tex, float x, float y) {return CUBICTEX2D<float3, short3, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<uint3, 2, mode> tex, float x, float y) {return CUBICTEX2D<float3, uint3, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<int3, 2, mode> tex, float x, float y) {return CUBICTEX2D<float3, int3, mode>(tex, x, y);}
// 4-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<float4, 2, mode> tex, float x, float y) {return CUBICTEX2D<float4, float4, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<uchar4, 2, mode> tex, float x, float y) {return CUBICTEX2D<float4, uchar4, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<char4, 2, mode> tex, float x, float y) {return CUBICTEX2D<float4, char4, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<ushort4, 2, mode> tex, float x, float y) {return CUBICTEX2D<float4, ushort4, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<short4, 2, mode> tex, float x, float y) {return CUBICTEX2D<float4, short4, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<uint4, 2, mode> tex, float x, float y) {return CUBICTEX2D<float4, uint4, mode>(tex, x, y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<int4, 2, mode> tex, float x, float y) {return CUBICTEX2D<float4, int4, mode>(tex, x, y);}

// Function definitions with (float2 coord) instead of (float x, float y)
// 1-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<float, 2, mode> tex, float2 coord) {return CUBICTEX2D<float, float, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<uchar, 2, mode> tex, float2 coord) {return CUBICTEX2D<float, uchar, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<char, 2, mode> tex, float2 coord) {return CUBICTEX2D<float, char, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<ushort, 2, mode> tex, float2 coord) {return CUBICTEX2D<float, ushort, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<short, 2, mode> tex, float2 coord) {return CUBICTEX2D<float, short, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<uint, 2, mode> tex, float2 coord) {return CUBICTEX2D<float, uint, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX2D(texture<int, 2, mode> tex, float2 coord) {return CUBICTEX2D<float, int, mode>(tex, coord.x, coord.y);}
// 2-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<float2, 2, mode> tex, float2 coord) {return CUBICTEX2D<float2, float2, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<uchar2, 2, mode> tex, float2 coord) {return CUBICTEX2D<float2, uchar2, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<char2, 2, mode> tex, float2 coord) {return CUBICTEX2D<float2, char2, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<ushort2, 2, mode> tex, float2 coord) {return CUBICTEX2D<float2, ushort2, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<short2, 2, mode> tex, float2 coord) {return CUBICTEX2D<float2, short2, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<uint2, 2, mode> tex, float2 coord) {return CUBICTEX2D<float2, uint2, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX2D(texture<int2, 2, mode> tex, float2 coord) {return CUBICTEX2D<float2, int2, mode>(tex, coord.x, coord.y);}
// 3-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<float3, 2, mode> tex, float2 coord) {return CUBICTEX2D<float3, float3, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<uchar3, 2, mode> tex, float2 coord) {return CUBICTEX2D<float3, uchar3, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<char3, 2, mode> tex, float2 coord) {return CUBICTEX2D<float3, char3, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<ushort3, 2, mode> tex, float2 coord) {return CUBICTEX2D<float3, ushort3, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<short3, 2, mode> tex, float2 coord) {return CUBICTEX2D<float3, short3, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<uint3, 2, mode> tex, float2 coord) {return CUBICTEX2D<float3, uint3, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX2D(texture<int3, 2, mode> tex, float2 coord) {return CUBICTEX2D<float3, int3, mode>(tex, coord.x, coord.y);}
// 4-dimensional pixels
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<float4, 2, mode> tex, float2 coord) {return CUBICTEX2D<float4, float4, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<uchar4, 2, mode> tex, float2 coord) {return CUBICTEX2D<float4, uchar4, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<char4, 2, mode> tex, float2 coord) {return CUBICTEX2D<float4, char4, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<ushort4, 2, mode> tex, float2 coord) {return CUBICTEX2D<float4, ushort4, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<short4, 2, mode> tex, float2 coord) {return CUBICTEX2D<float4, short4, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<uint4, 2, mode> tex, float2 coord) {return CUBICTEX2D<float4, uint4, mode>(tex, coord.x, coord.y);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX2D(texture<int4, 2, mode> tex, float2 coord) {return CUBICTEX2D<float4, int4, mode>(tex, coord.x, coord.y);}

