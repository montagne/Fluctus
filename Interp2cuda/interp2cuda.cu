#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "cutil.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


#include <memcpy.cu>
#include <cubicPrefilter2D.cu>
#include <cubicTex2D.cu>


#define IMUL(a, b) __umul24(a, b)

//////////////////////////////////////////////////////////////////////////////////////
///         Interpolation Kernel
//////////////////////////////////////////////////////////////////////////////////////
texture<float, 2, hipReadModeElementType> texInput;

__global__ void interpTex( float2 *points, float *output, int output_numel, int output_width, int _option){
  const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
  const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	
  const int loc = IMUL(y,output_width) + x;
  if (loc < output_numel) { 

    switch(_option){
    case 0:
      output[loc]  = tex2D(texInput, points[loc].x, points[loc].y);
      break;
    case 1:
      output[loc]  = tex2D(texInput, points[loc].x, points[loc].y);
      break;
    case 2:
      output[loc]  = cubicTex2D(texInput, points[loc].x, points[loc].y);
      break;
    case 3:
      output[loc]  = cubicTex2D(texInput, points[loc].x, points[loc].y);
      break;
    default :
      output[loc]  = tex2D(texInput, points[loc].x, points[loc].y);
      break;
    }

  }
}



//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
  return (a % b != 0) ? (a / b + 1) : (a / b);
}

//////////////////////////////////////////////////////////////////////////////////////
///         Main
//////////////////////////////////////////////////////////////////////////////////////

void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] ) {
  // Check number of inputs
  if (nrhs <3) mexErrMsgTxt("Must have three input arguments: data, rowpoints, colpoints");
  if (nlhs !=1) mexErrMsgTxt("Must have one output argument");

  if (nrhs > 4) mexErrMsgTxt("This format is not supported, usage : interp2cuda(W,Xi,Yi,method),  method is optional ");
	
  // Check the class of input data
  if ( mxIsComplex(prhs[0]) || !mxIsClass(prhs[0],"double") ) {
    // try casting the input to double?
    mexErrMsgTxt("Input must be real, double !");
  }
	
  ///// Allocate, set up data structures
  int OUTPUT_W, OUTPUT_H, OUTPUT_SIZE, INPUT_W, INPUT_H, INPUT_SIZE, POINTS_SIZE ;

  int interpo_option;	

  double *input = mxGetPr(prhs[0]);


  INPUT_W = mxGetN(prhs[0]);
  INPUT_H = mxGetM(prhs[0]);

  INPUT_SIZE = INPUT_H * INPUT_W * sizeof(float);

  float *f_input;
  float2 *f_points;
	
  ///// Check if we're in "input points" or "number of points" mode
  if ( mxGetNumberOfElements(prhs[1]) == 1 && mxGetNumberOfElements(prhs[2]) == 1) {

    mexErrMsgTxt("This mode is not yet implemented !! usage interp2(Z,Xi,Yi,method)");

    // double *row_points = mxGetPr(prhs[1]);
    // double *col_points = mxGetPr(prhs[2]);


    // // number of points mode
    // OUTPUT_W = (int) col_points[0];
    // OUTPUT_H = (int) row_points[0];
    // OUTPUT_SIZE = OUTPUT_W * OUTPUT_H * sizeof(float);
    // POINTS_SIZE = 2*OUTPUT_SIZE;

		
    // // we want N evenly spaced points from 0 to 1
    // f_points = (float2 *)mxMalloc(POINTS_SIZE);
		
    // for ( int r=0; r<OUTPUT_H; r++ ) {
    //   for ( int c=0; c<OUTPUT_W; c++ ) {
    // 	f_points[c + OUTPUT_W*r].x = (float) c * (INPUT_W-1) / (OUTPUT_W-1) + 0.5f;
    // 	f_points[c + OUTPUT_W*r].y = (float) r * (INPUT_H-1) / (OUTPUT_H-1) + 0.5f;
    //   }
    // }
  }
  else {
    double *x_points = mxGetPr(prhs[1]);
    double *y_points = mxGetPr(prhs[2]);
    int NB_ELEM_Xi,NB_ELEM_Yi;

    if (nrhs == 4 || nrhs == 7){ // if interpolation option is passed 

      mxChar* option = (nrhs == 4) ? mxGetChars(prhs[3]) : mxGetChars(prhs[6]);

      if((char)option[0] == 'n' && (char)option[1] == 'e' && (char)option[2] == 'a' && (char)option[3] == 'r' && (char)option[4] == 'e' 
	 && (char)option[5] == 's' && (char)option[6] == 't'){
	interpo_option = 0;
	//mexPrintf("nearest\n");
      }
      else if ((char)option[0] == 'l' && (char)option[1] == 'i' && (char)option[2] == 'n' && (char)option[3] == 'e' && (char)option[4] == 'a' 
	       && (char)option[5] == 'r'){
	interpo_option = 1;
	//mexPrintf("linear\n");
      }
      else if((char)option[0] == 's' && (char)option[1] == 'p' && (char)option[2] == 'l' && (char)option[3] == 'i' && (char)option[4] == 'n' 
	      && (char)option[5] == 'e'){
	interpo_option = 2;
	//mexPrintf("spline\n");
      }
      else if((char)option[0] == 'c' && (char)option[1] == 'u' && (char)option[2] == 'b' && (char)option[3] == 'i' && (char)option[4] == 'c'){
	interpo_option = 3;
	//mexPrintf("cubic\n");
	mexErrMsgTxt("cubic method is not yet supported !!");
      }
      else{
	mexErrMsgTxt("method is not recognized, you must use 'nearest' 'linear' 'spline' or 'cubic'");
      }

    }
    else{
      interpo_option = 1;
      //      mexPrintf("linear\n"); 
    }



    NB_ELEM_Xi = mxGetNumberOfElements(prhs[1]);
    NB_ELEM_Yi = mxGetNumberOfElements(prhs[2]);

    if ( NB_ELEM_Xi != NB_ELEM_Yi){
      mexErrMsgTxt("Xi and Yi must have the same size !!");
    }

    OUTPUT_W = mxGetN(prhs[1]);
    OUTPUT_H = mxGetM(prhs[1]);
    OUTPUT_SIZE = OUTPUT_W * OUTPUT_H * sizeof(float);
    POINTS_SIZE = 2*OUTPUT_SIZE;

    f_points = (float2 *)mxMalloc(POINTS_SIZE);
		
    for ( int r=0; r<NB_ELEM_Xi; r++ ) {
      f_points[r].x = (float) x_points[r] + 0.5f;
      f_points[r].y = (float) y_points[r] + 0.5f;
    }


  }
	
  plhs[0] = mxCreateDoubleMatrix(OUTPUT_H, OUTPUT_W, mxREAL);
  double *output = mxGetPr(plhs[0]);
  float *f_output = (float *)mxMalloc(OUTPUT_SIZE);
	


  ///// We need to convert the input array from double to float
  f_input = (float *)malloc(INPUT_SIZE);

  for ( int r=0; r<INPUT_H; r++ ) {
    for ( int c=0; c<INPUT_W; c++ ) {
      f_input[c + INPUT_W*r] = (float) input[r+ (INPUT_H)*(c)];
    }
  }

  // --- CUDA Part ---
	
  // Allocate, copy input data into a 2D texture
  hipArray *d_input;
  hipChannelFormatDesc input_tex = hipCreateChannelDesc<float>();
	

  hipMallocArray(&d_input, &input_tex, INPUT_W, INPUT_H);
  hipMemcpyToArray(d_input, 0, 0, f_input, INPUT_SIZE, hipMemcpyHostToDevice);

  texInput.filterMode = (interpo_option == 0 ) ? hipFilterModePoint : hipFilterModeLinear;
  texInput.normalized = 0;
	
  hipBindTextureToArray(texInput, d_input);
	
  // Allocate, copy points data into a float2*
  float2 *d_points;
  hipMalloc((void **)&d_points, POINTS_SIZE);
  hipMemcpy(d_points, f_points, POINTS_SIZE, hipMemcpyHostToDevice);
	
  // Allocate output space
  float *d_output;
  hipMalloc((void **)&d_output, OUTPUT_SIZE);
	
  // Set up blocks, grid for parallel processing
  dim3 dimBlock(16, 12);
  dim3 dimGrid(iDivUp(OUTPUT_W,dimBlock.x),iDivUp(OUTPUT_H,dimBlock.y));
	
  // Run it
  interpTex<<<dimGrid, dimBlock>>> (d_points, d_output, OUTPUT_W*OUTPUT_H, OUTPUT_W, interpo_option);

  // Copy the data back
  hipMemcpy(f_output, d_output, OUTPUT_SIZE, hipMemcpyDeviceToHost);
		
  // Convert to double, and we're done
  for ( int r=0; r<OUTPUT_H; r++ ) {
    for ( int c=0; c<OUTPUT_W; c++ ) {
      output[r+OUTPUT_H*c] = (double)f_output[r+OUTPUT_H*c];
    }
  }

				
  hipUnbindTexture(texInput);
  hipFreeArray(d_input);
	
  hipFree(d_points);
  hipFree(d_output);
	
  mxFree(f_input);
  mxFree(f_output);
  mxFree(f_points);
}
