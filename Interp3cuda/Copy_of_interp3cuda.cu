#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "cutil.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdio.h>
#include <stdlib.h>


#include <memcpy.cu>
#include <cubicPrefilter3D.cu>
#include <cubicTex3D.cu>


//static bool debug = false;

#define IMUL(a, b) __umul24(a, b)



//////////////////////////////////////////////////////////////////////////////////////
///         Interpolation Kernel
//////////////////////////////////////////////////////////////////////////////////////
texture<float, 3, hipReadModeElementType> texInput;

__global__ void interpTex( float3 *points, float *output, int output_numel, int output_width, int _option){

  const uint x = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  const uint y = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;

  uint loc = IMUL(y, output_width) + x;
  if (loc < output_numel) { 
    switch(_option){
    case 0:
      output[loc]  = tex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    case 1:
      output[loc]  = tex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    case 2:
      output[loc]  = cubicTex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    case 3:
      output[loc]  = cubicTex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    default :
      output[loc]  = tex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    }

  }


}



//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
  return (a % b != 0) ? (a / b + 1) : (a / b);
}

//////////////////////////////////////////////////////////////////////////////////////
///         Main
//////////////////////////////////////////////////////////////////////////////////////

void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] ) {
  // Check number of inputs
  if (nrhs <3) mexErrMsgTxt("Must have three input arguments: data, rowpoints, colpoints");

  if (nrhs > 5) mexErrMsgTxt("This format is not supported, usage : interp3cuda(W,Xi,Yi,Zi,method),  method is optional ");
  if (nlhs !=1) mexErrMsgTxt("Must have one output argument");
	
  // Check the class of input data
  if ( mxIsComplex(prhs[0]) || !mxIsClass(prhs[0],"double") ) {
    // try casting the input to double?
    mexErrMsgTxt("Input must be real, double !");
  }
	

  ///// Allocate, set up data structures
  int OUTPUT_W, OUTPUT_H, OUTPUT_P, OUTPUT_SIZE;
  int INPUT_W, INPUT_H,INPUT_P, INPUT_SIZE, POINTS_SIZE;

  int NB_ELEM_Xi,NB_ELEM_Yi,NB_ELEM_Zi;	

  int interpo_option;

  double *input = mxGetPr(prhs[0]);

  const mwSize* dim_input = mxGetDimensions(prhs[0]);

  INPUT_W = dim_input[0];
  INPUT_H = dim_input[1];
  INPUT_P = dim_input[2];

  //  mexPrintf("W %d  H %d  P %d\n",INPUT_W, INPUT_H, INPUT_P);

  INPUT_SIZE = INPUT_H * INPUT_W * INPUT_P * sizeof(float);


  float *f_input;
  float3 *f_points;
	
  ///// Check if we're in "input points" or "number of points" mode
  if ( mxGetNumberOfElements(prhs[1]) == 1 && mxGetNumberOfElements(prhs[2]) == 1) {

    //  mexPrintf("number of point\n");

    double *row_points = mxGetPr(prhs[1]);
    double *col_points = mxGetPr(prhs[2]);


    // number of points mode
    OUTPUT_W = (int) col_points[0];
    OUTPUT_H = (int) row_points[0];
    OUTPUT_SIZE = OUTPUT_W * OUTPUT_H * sizeof(float);
    POINTS_SIZE = 2*OUTPUT_SIZE;

		
    // we want N evenly spaced points from 0 to 1
    f_points = (float3 *)mxMalloc(POINTS_SIZE);
		
    for ( int r=0; r<OUTPUT_H; r++ ) {
      for ( int c=0; c<OUTPUT_W; c++ ) {
	f_points[c + OUTPUT_W*r].x = (float) c * (INPUT_W-1) / (OUTPUT_W-1) + 0.5f;
	f_points[c + OUTPUT_W*r].y = (float) r * (INPUT_H-1) / (OUTPUT_H-1) + 0.5f;
      }
    }

  }
  else {

    //    mexPrintf("my test\n");

    double *x_points = mxGetPr(prhs[1]);
    double *y_points = mxGetPr(prhs[2]);
    double *z_points = mxGetPr(prhs[3]);

    if (nrhs == 5 || nrhs == 8){ // if interpolation option is passed 
      //FIXME : the condition must be : if nrhs == 5 

      mxChar* option = (nrhs == 5) ? mxGetChars(prhs[4]) : mxGetChars(prhs[7]);

      if((char)option[0] == 'n' && (char)option[1] == 'e' && (char)option[2] == 'a' && (char)option[3] == 'r' && (char)option[4] == 'e' 
	 && (char)option[5] == 's' && (char)option[6] == 't'){
	interpo_option = 0;
	//	mexPrintf("nearest\n");
      }
      else if ((char)option[0] == 'l' && (char)option[1] == 'i' && (char)option[2] == 'n' && (char)option[3] == 'e' && (char)option[4] == 'a' 
	       && (char)option[5] == 'r'){
	interpo_option = 1;
	//	mexPrintf("linear\n");
      }
      else if((char)option[0] == 's' && (char)option[1] == 'p' && (char)option[2] == 'l' && (char)option[3] == 'i' && (char)option[4] == 'n' 
	      && (char)option[5] == 'e'){
	interpo_option = 2;
	//	mexPrintf("spline\n");
      }
      else if((char)option[0] == 'c' && (char)option[1] == 'u' && (char)option[2] == 'b' && (char)option[3] == 'i' && (char)option[4] == 'c'){
	interpo_option = 3;
	//	mexPrintf("cubic\n");
	mexErrMsgTxt("cubic method is not yet supported !!");
      }
      else{
	mexErrMsgTxt("method is not recognized, you must use 'nearest' 'linear' 'spline' or 'cubic'");
      }

    }
    else{
      interpo_option = 1;
      //      mexPrintf("linear\n"); 
    }
    
    NB_ELEM_Xi = mxGetNumberOfElements(prhs[1]);
    NB_ELEM_Yi = mxGetNumberOfElements(prhs[2]);
    NB_ELEM_Zi = mxGetNumberOfElements(prhs[3]);

    if ( (NB_ELEM_Xi != NB_ELEM_Yi) && (NB_ELEM_Yi != NB_ELEM_Zi)){
      mexErrMsgTxt("Xi, Yi and Zi must have the same number of element  !!");
    }


    const mwSize* dimX = mxGetDimensions(prhs[1]);
    const mwSize* dimY = mxGetDimensions(prhs[2]);
    const mwSize* dimZ = mxGetDimensions(prhs[3]);


    if (dimX[0] != dimY[0] || dimY[0] != dimZ[0] || dimZ[0] != dimX[0] || dimX[1] != dimY[1] || dimY[1] != dimZ[1] || dimZ[1] != dimX[1]
	|| dimX[2] != dimY[2] || dimY[2] != dimZ[2] || dimZ[2] != dimX[2]){
          mexErrMsgTxt("Xi, Yi and Zi dimension must be the same !!");
    }

  plhs[0] = mxCreateNumericArray(3, dimX, mxDOUBLE_CLASS ,mxREAL);

    OUTPUT_W = dimX[0];
    OUTPUT_H = dimX[1];
    OUTPUT_P = dimX[2];

    OUTPUT_SIZE = OUTPUT_W * OUTPUT_H * OUTPUT_P * sizeof(float);
    POINTS_SIZE = 3*OUTPUT_SIZE;



    f_points = (float3 *)mxMalloc(POINTS_SIZE);
		
    for ( int r=0; r<NB_ELEM_Xi; r++ ) {
      f_points[r].x = (float) x_points[r] - 0.5f;
      f_points[r].y = (float) y_points[r] - 0.5f;
      f_points[r].z = (float) z_points[r] - 0.5f;
    }

   }
	


   double *output = mxGetPr(plhs[0]);
   float *f_output = (float *)mxMalloc(OUTPUT_SIZE);
	


  ///// We need to convert the input array from double to float
  f_input = (float *)malloc(INPUT_SIZE);


  for ( int p = 0; p < INPUT_P; p++ ) {
    for ( int c = 0; c < INPUT_W; c++ ) {
      for ( int r = 0; r < INPUT_H; r++){
  	f_input[(c + INPUT_W*r + p*INPUT_W*INPUT_H)] = (float) input[r+ (INPUT_H)*c + p*INPUT_W*INPUT_H];
      }
    }
  }

  ///// Load all this stuff into graphics memory
	
  // Allocate, copy input data into a 2D texture
  hipArray *d_input;
  hipChannelFormatDesc input_tex = hipCreateChannelDesc<float>();

  const hipExtent volumeSize = make_hipExtent(OUTPUT_W, OUTPUT_H, OUTPUT_P);
	
  cutilSafeCall( hipMalloc3DArray(&d_input, &input_tex, volumeSize) );

  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr   = make_hipPitchedPtr((void*)f_input, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
  copyParams.dstArray = d_input;
  copyParams.extent   = volumeSize;
  copyParams.kind     = hipMemcpyHostToDevice;

  hipMemcpy3D(&copyParams) ;



  texInput.filterMode = (interpo_option == 0 ) ? hipFilterModePoint : hipFilterModeLinear;
  texInput.normalized = 0;
  texInput.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
  texInput.addressMode[1] = hipAddressModeWrap;
  texInput.addressMode[2] = hipAddressModeWrap;

  // bind array to 3D texture
  hipBindTextureToArray(texInput, d_input, input_tex);


  // Allocate, copy points data into a float2*
  float3 *d_points;
  hipMalloc((void **)&d_points, POINTS_SIZE);
  hipMemcpy(d_points, f_points, POINTS_SIZE, hipMemcpyHostToDevice);

	
  // Allocate output space
  float *d_output;
  hipMalloc((void **)&d_output, OUTPUT_SIZE);
	
  const dim3 blockSize(min(PowTwoDivider(OUTPUT_W), 16), min(PowTwoDivider(OUTPUT_H * OUTPUT_P), 16));
  const dim3 gridSize(OUTPUT_W/ blockSize.x, OUTPUT_H * OUTPUT_P / blockSize.y);


  // Run it
  interpTex<<<gridSize, blockSize>>> (d_points, d_output, OUTPUT_W*OUTPUT_H*OUTPUT_P, OUTPUT_W, interpo_option);


  // Copy the data back
  hipMemcpy(f_output, d_output, OUTPUT_SIZE, hipMemcpyDeviceToHost);


  // Convert to double, and we're done
  for ( int r = 0; r<OUTPUT_H; r++ ) {
    for ( int c = 0; c<OUTPUT_W; c++ ) {
      for(int p = 0; p < OUTPUT_P; p++){
	output[r + OUTPUT_H*c + p*OUTPUT_H*OUTPUT_W] = (double)f_output[r+OUTPUT_H*c +p*OUTPUT_H*OUTPUT_W];
      }
    }
  }

  // mexPrintf("f_output = \n");
  // for (int i = 0; i < 8; i++) {
  //   mexPrintf("[%d] %f \n",i,f_output[i]);
  // }
  // mexPrintf("\n");

  //  mexPrintf("f_output [0] :%d\n",(int)f_output[0]);

				
  hipUnbindTexture(texInput);
  hipFreeArray(d_input);
	
  hipFree(d_points);
  hipFree(d_output);
	
  mxFree(f_input);
  mxFree(f_output);
  mxFree(f_points);
}
