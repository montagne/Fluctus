#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "cutil.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <stdio.h>
#include <stdlib.h>


#include <memcpy.cu>
#include <cubicPrefilter3D.cu>
#include <cubicTex3D.cu>


#define IMUL(a, b) __umul24(a, b)



//////////////////////////////////////////////////////////////////////////////////////
///         Interpolation Kernel
//////////////////////////////////////////////////////////////////////////////////////
texture<float, 3, hipReadModeElementType> texInput;

__global__ void interpTex( float3 *points, float *output, int output_numel, int output_width, int _option){

  const uint x = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  const uint y = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;

  uint loc = IMUL(y, output_width) + x;
  if (loc < output_numel) { 
    switch(_option){
    case 0:
      //nearest mode point
      output[loc]  = tex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    case 1:
      //linear mode linear
      output[loc]  = tex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    case 2:
      // spline mode linear
      output[loc]  = cubicTex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    case 3:
      // cubic mode linear
      output[loc]  = cubicTex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    default :
      //linear mode linear
      output[loc]  = tex3D(texInput, points[loc].x, points[loc].y, points[loc].z);
      break;
    }

  }


}



//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
  return (a % b != 0) ? (a / b + 1) : (a / b);
}


int option_method( int nrhs, const mxArray *prhs[]) {

  int interpo_option;  
    if (nrhs == 5 || nrhs == 8){ // if interpolation option is passed  

      mxChar* option = (nrhs == 5) ? mxGetChars(prhs[4]) : mxGetChars(prhs[7]);

      if((char)option[0] == 'n' && (char)option[1] == 'e' && (char)option[2] == 'a' && (char)option[3] == 'r' && (char)option[4] == 'e' 
	 && (char)option[5] == 's' && (char)option[6] == 't'){
	interpo_option = 0;
	//	mexPrintf("nearest\n");
      }
      else if ((char)option[0] == 'l' && (char)option[1] == 'i' && (char)option[2] == 'n' && (char)option[3] == 'e' && (char)option[4] == 'a' 
	       && (char)option[5] == 'r'){
	interpo_option = 1;
	//	mexPrintf("linear\n");
      }
      else if((char)option[0] == 's' && (char)option[1] == 'p' && (char)option[2] == 'l' && (char)option[3] == 'i' && (char)option[4] == 'n' 
	      && (char)option[5] == 'e'){
	interpo_option = 2;
	//	mexPrintf("spline\n");
      }
      else if((char)option[0] == 'c' && (char)option[1] == 'u' && (char)option[2] == 'b' && (char)option[3] == 'i' && (char)option[4] == 'c'){
	interpo_option = 3;
	//	mexPrintf("cubic\n");
	mexErrMsgTxt("cubic method is not yet supported !!");
      }
      else{
	mexErrMsgTxt("method is not recognized, you must use 'nearest' 'linear' 'spline' or 'cubic'");
      }

    }
    else{
      interpo_option = 1;
      //      mexPrintf("linear\n"); 
    }
  
    return interpo_option;
  
}


//////////////////////////////////////////////////////////////////////////////////////
///         Main
//////////////////////////////////////////////////////////////////////////////////////

void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] ) {
  // Check number of inputs
  if (nrhs <3) mexErrMsgTxt("Must have three input arguments: data, rowpoints, colpoints");

  if (nrhs > 5) mexErrMsgTxt("This format is not supported, usage : interp3cuda(W,Xi,Yi,Zi,method),  method is optional ");
  if (nlhs !=1) mexErrMsgTxt("Must have one output argument");
	
  // Check the class of input data
  if ( mxIsComplex(prhs[0]) || !mxIsClass(prhs[0],"double") ) {
    // try casting the input to double?
    mexErrMsgTxt("Input must be real, double !");
  }
	

  ///// Allocate, set up data structures
  int OUTPUT_W, OUTPUT_H, OUTPUT_D, OUTPUT_SIZE;
  int INPUT_W, INPUT_H,INPUT_D, INPUT_SIZE, POINTS_SIZE;

  int NB_ELEM_Xi,NB_ELEM_Yi,NB_ELEM_Zi;	

  int interpo_option;

  double *input = mxGetPr(prhs[0]);

  const mwSize* dim_input = mxGetDimensions(prhs[0]);

  INPUT_W = dim_input[0];
  INPUT_H = dim_input[1];
  INPUT_D = dim_input[2];

  INPUT_SIZE = INPUT_H * INPUT_W * INPUT_D * sizeof(float);


  float *f_input;
  float3 *f_points;
	
  ///// Check if we're in "input points" or "number of points" mode
  if ( mxGetNumberOfElements(prhs[1]) == 1 && mxGetNumberOfElements(prhs[2]) == 1) {

    //  mexPrintf("number of point\n");

    double *row_points = mxGetPr(prhs[1]);
    double *col_points = mxGetPr(prhs[2]);


    // number of points mode
    OUTPUT_W = (int) col_points[0];
    OUTPUT_H = (int) row_points[0];
    OUTPUT_SIZE = OUTPUT_W * OUTPUT_H * sizeof(float);
    POINTS_SIZE = 2*OUTPUT_SIZE;

		
    // we want N evenly spaced points from 0 to 1
    f_points = (float3 *)mxMalloc(POINTS_SIZE);
		
    for ( int r=0; r<OUTPUT_H; r++ ) {
      for ( int c=0; c<OUTPUT_W; c++ ) {
        f_points[c + OUTPUT_W*r].x = (float) c * (INPUT_W-1) / (OUTPUT_W-1) + 0.5f;
        f_points[c + OUTPUT_W*r].y = (float) r * (INPUT_H-1) / (OUTPUT_H-1) + 0.5f;
      }
    }

  }
  else {

    double *x_points = mxGetPr(prhs[1]);
    double *y_points = mxGetPr(prhs[2]);
    double *z_points = mxGetPr(prhs[3]);


    interpo_option = option_method (nrhs, prhs);

    
    NB_ELEM_Xi = mxGetNumberOfElements(prhs[1]);
    NB_ELEM_Yi = mxGetNumberOfElements(prhs[2]);
    NB_ELEM_Zi = mxGetNumberOfElements(prhs[3]);

    if ( (NB_ELEM_Xi != NB_ELEM_Yi) && (NB_ELEM_Yi != NB_ELEM_Zi)){
      mexErrMsgTxt("Xi, Yi and Zi must have the same number of element  !!");
    }


    const mwSize* dimX = mxGetDimensions(prhs[1]);
    const mwSize* dimY = mxGetDimensions(prhs[2]);
    const mwSize* dimZ = mxGetDimensions(prhs[3]);


    if (dimX[0] != dimY[0] || dimY[0] != dimZ[0] || dimZ[0] != dimX[0] || dimX[1] != dimY[1] || dimY[1] != dimZ[1] || dimZ[1] != dimX[1]
	|| dimX[2] != dimY[2] || dimY[2] != dimZ[2] || dimZ[2] != dimX[2]){
          mexErrMsgTxt("Xi, Yi and Zi dimension must be the same !!");
    }

    plhs[0] = mxCreateNumericArray(3, dimX, mxDOUBLE_CLASS ,mxREAL);

    OUTPUT_W = dimX[0];
    OUTPUT_H = dimX[1];
    OUTPUT_D = dimX[2];
    OUTPUT_SIZE = OUTPUT_W * OUTPUT_H * OUTPUT_D * sizeof(float);
    POINTS_SIZE = 3*OUTPUT_SIZE;

    f_points = (float3 *)mxMalloc(POINTS_SIZE);
		
    for ( int r=0; r<NB_ELEM_Xi; r++ ) {
      f_points[r].x = (float) x_points[r] - 0.5f;
      f_points[r].y = (float) y_points[r] - 0.5f;
      f_points[r].z = (float) z_points[r] - 0.5f;
    }
   }

   double *output = mxGetPr(plhs[0]);
   float *f_output = (float *)mxMalloc(OUTPUT_SIZE);
	

  // We need to convert the input array from double to float
  f_input = (float *)malloc(INPUT_SIZE);

   
  for ( int d = 0; d < INPUT_D; d++ ) {
    for ( int r = 0; r < INPUT_H; r++){
      for ( int c = 0; c < INPUT_W; c++ ) {
	f_input[(c + INPUT_W*r + d*INPUT_W*INPUT_H)] = (float) input[c + INPUT_W*r + d*INPUT_W*INPUT_H];
      }
    }
  }

	
  //---------------------------
  // ------ CUDA Part ---------
  //---------------------------

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess) {
    mexPrintf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
    mexErrMsgTxt("CUDA device not found");
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0); //only on device #0

  if (INPUT_W > deviceProp.maxTexture3D[0] || INPUT_H > deviceProp.maxTexture3D[1] || INPUT_D > deviceProp.maxTexture3D[2]){
    mexPrintf("One of input dimension is greater than CUDA capabilites\n");
    mexPrintf("Max dimension are : (%d,%d,%d)\n",deviceProp.maxTexture3D[0],deviceProp.maxTexture3D[1],deviceProp.maxTexture3D[2]);
    mexErrMsgTxt("ERROR !!");
  }





  hipArray *d_input = 0;
  const hipExtent volumeExtent = make_hipExtent(INPUT_W, INPUT_H, INPUT_D);

  //  CreateTextureFromVolume(&texInput, &d_input, f_input, volumeExtent, false);


  hipChannelFormatDesc input_tex = hipCreateChannelDesc<float>();
  cutilSafeCall( hipMalloc3DArray(&d_input, &input_tex, volumeExtent) );

  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr   = make_hipPitchedPtr((void*)f_input, volumeExtent.width * sizeof(float), volumeExtent.width, volumeExtent.height);
  copyParams.dstArray = d_input;
  copyParams.extent   = volumeExtent;
  copyParams.kind     = hipMemcpyHostToDevice;

  cutilSafeCall( hipMemcpy3D(&copyParams) );

  texInput.filterMode = (interpo_option == 0 ) ? hipFilterModePoint : hipFilterModeLinear;
  texInput.normalized = 0;
  texInput.addressMode[0] = hipAddressModeClamp;
  texInput.addressMode[1] = hipAddressModeClamp;
  texInput.addressMode[2] = hipAddressModeClamp;

  // bind array to 3D texture
  cutilSafeCall(  hipBindTextureToArray(texInput, d_input, input_tex));


  // Allocate, copy points data into a float3*
  float3 *d_points;
  cutilSafeCall( hipMalloc((void **)&d_points, POINTS_SIZE));
  cutilSafeCall( hipMemcpy(d_points, f_points, POINTS_SIZE, hipMemcpyHostToDevice));

	
  // Allocate output space
  float *d_output;
  cutilSafeCall(  hipMalloc((void **)&d_output, OUTPUT_SIZE));
	
  const dim3 blockSize(min(PowTwoDivider(OUTPUT_W), 16), min(PowTwoDivider(OUTPUT_H * OUTPUT_D), 16));
  const dim3 gridSize(OUTPUT_W/ blockSize.x, OUTPUT_H * OUTPUT_D / blockSize.y);


  // Run it
  interpTex<<<gridSize, blockSize>>> (d_points, d_output, OUTPUT_W*OUTPUT_H*OUTPUT_D, OUTPUT_W, interpo_option);


  // Copy the data back
  cutilSafeCall(  hipMemcpy(f_output, d_output, OUTPUT_SIZE, hipMemcpyDeviceToHost));


  // Convert to double, and we're done
  for ( int r = 0; r<OUTPUT_H; r++ ) {
    for ( int c = 0; c<OUTPUT_W; c++ ) {
      for( int d = 0; d < OUTPUT_D; d++){
	output[r + OUTPUT_H*c + d*OUTPUT_H*OUTPUT_W] = (double)f_output[r + OUTPUT_H*c + d*OUTPUT_H*OUTPUT_W];
      }
    }
  }


				
  cutilSafeCall( hipUnbindTexture(texInput));
  cutilSafeCall( hipFreeArray(d_input)) ;
	
  cutilSafeCall( hipFree(d_points));
  cutilSafeCall( hipFree(d_output));
	
  mxFree(f_input);
  mxFree(f_output);
  mxFree(f_points);
}
